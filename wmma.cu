
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>

using namespace nvcuda;

__global__ void wmma_ker(half *a, half *b, float *c) {
  // Declare the fragments
  wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::col_major> a_frag;
  wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
  wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;

  // Initialize the output to zero
  wmma::fill_fragment(c_frag, 0.0f);

  // Load the inputs
  // 16 is the leading dimension
  wmma::load_matrix_sync(a_frag, a, 16);
  wmma::load_matrix_sync(b_frag, b, 16);

  // Perform the matrix multiplication
  wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

  // Store the output
  wmma::store_matrix_sync(c, c_frag, 16, wmma::mem_row_major);
}

int main() {
  half *d_a, *h_a, *d_b, *h_b;
  float *d_c, *h_c;
  h_c = new float[16 * 16];
  h_b = new half[16 * 16];
  h_a = new half[16 * 16];
  hipMalloc(&d_a, 16 * 16 * sizeof(half));
  hipMalloc(&d_b, 16 * 16 * sizeof(half));
  hipMalloc(&d_c, 16 * 16 * sizeof(float));
  for (int i = 0; i < 16 * 16; i++) {
    h_a[i] = 1.0f;
    h_b[i] = 1.0f;
  }
  hipMemcpy(d_a, h_a, 16 * 16 * sizeof(half), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, 16 * 16 * sizeof(half), hipMemcpyHostToDevice);
  wmma_ker<<<1, 32>>>(d_a, d_b, d_c);
  hipMemcpy(h_c, d_c, 16 * 16 * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < 16 * 16; i++)
    std::cout << h_c[i] << ",";
  std::cout << std::endl;
}
