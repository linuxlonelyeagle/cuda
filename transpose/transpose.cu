#include <hip/hip_runtime.h>
#include <iostream>

// read row, write col;
__global__ void transpose0(float *in, float *out, int nx, int ny) {
  int ix = blockDim.x * blockIdx.x + threadIdx.x;
  int iy = blockDim.y * blockIdx.y + threadIdx.y;
  out[iy + ix * ny] = in[ix + iy * nx];
}

// read col, write row;
__global__ void transpose1(float *in, float *out, int nx, int ny) {
  int ix = blockDim.x * blockIdx.x + threadIdx.x;
  int iy = blockDim.y * blockIdx.y + threadIdx.y;
  out[ix + iy * nx] = in[iy + ix * ny];
}

__global__ void transposeOptimized(float *input, float *output, int m, int n) {
  int colID_input = threadIdx.x + blockDim.x * blockIdx.x;
  int rowID_input = threadIdx.y + blockDim.y * blockIdx.y;
  __shared__ float sdata[32][33];
  if (colID_input < m && rowID_input < n) {
    int index_input = colID_input + rowID_input * n;
    sdata[threadIdx.y][threadIdx.x] = input[index_input];
    __syncthreads();
    int dst_col = threadIdx.x + blockIdx.y * blockDim.y;
    int dst_row = threadIdx.y + blockIdx.x * blockDim.x;
    output[dst_col + dst_row * m] = sdata[threadIdx.x][threadIdx.y];
  }
}

int main(int argc, char *argv[]) {
  int nx = 1 << 12;
  int ny = 1 << 12;
  size_t bytes = nx * ny * sizeof(float);
  float *h_in = (float *)malloc(bytes);
  float *h_out = (float *)malloc(bytes);
  float *d_in, *d_out;
  hipMalloc(&d_in, bytes);
  hipMalloc(&d_out, bytes);
  // init data;
  for (int i = 0; i < nx * ny; ++i) {
    h_in[i] = i;
  }
  hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice);
  hipEvent_t start;
  hipEvent_t stop;
  float msec;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  {
    dim3 block(32, 16);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    hipEventRecord(start);
    for (int i = 0; i < 100; ++i) {
      transpose0<<<grid, block>>>(d_in, d_out, nx, ny);
    }
    hipEventRecord(stop);
    hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost);
    hipEventElapsedTime(&msec, start, stop);
    printf("read row, write col\n");
    printf("transpose0 takes %.3f msec\n", msec / 100);
  }
  {
    dim3 block(32, 16);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    hipEventRecord(start);
    for (int i = 0; i < 100; ++i) {
      transpose1<<<grid, block>>>(d_in, d_out, nx, ny);
    }
    hipEventRecord(stop);
    hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost);
    hipEventElapsedTime(&msec, start, stop);
    printf("read col, write row\n");
    printf("transpose1 takes %.3f msec\n", msec / 100);
  }
  {
    dim3 block(32, 32);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    hipEventRecord(start);
    for (int i = 0; i < 100; ++i) {
      transposeOptimized<<<grid, block>>>(d_in, d_out, nx, ny);
    }
    hipEventRecord(stop);
    hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost);
    hipEventElapsedTime(&msec, start, stop);
    printf("transposeOptimized takes %.3f msec\n", msec / 100);
    /*
    for (int i = 0; i < ny; ++i) {
      for (int j = 0; j < nx; ++j) {
        int idx = i * nx + j;
        printf("%f ", h_out[idx]);
      }
      printf("\n");
    }
    printf("\n");
    */
  }
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(d_in);
  hipFree(d_out);
  free(h_in);
  free(h_out);
  return 0;
}
