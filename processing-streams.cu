#include "hip/hip_runtime.h"
#include <cstdio>

#define N 32

__global__ void kernel(int *a, int *b, int *c) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < N) {
    c[idx] = a[idx] + b[idx];
  }
}

int main(int argc, char *argv[]) {
  int *host_a = nullptr, *host_b = nullptr, *host_c = nullptr;
  int *dev_a = nullptr, *dev_b = nullptr, *dev_c = nullptr;
  int dataSize = N * sizeof(int);
  hipStream_t stream;
  hipStreamCreate(&stream);

  // allocate some page-locked memory
  hipHostAlloc((void **)&host_a, dataSize, hipHostMallocDefault);
  hipHostAlloc((void **)&host_b, dataSize, hipHostMallocDefault);
  hipHostAlloc((void **)&host_c, dataSize, hipHostMallocDefault);
  for (int i = 0; i < N; ++i) {
    host_a[i] = i;
    host_b[i] = i;
  }

  // alloc gpu memrey.
  hipMalloc((void **)&dev_a, dataSize);
  hipMalloc((void **)&dev_b, dataSize);
  hipMalloc((void **)&dev_c, dataSize);

  // copy host memrey to gpu memrey.
  hipMemcpyAsync(dev_a, host_a, dataSize, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(dev_b, host_b, dataSize, hipMemcpyHostToDevice, stream);

  // asynchronous calls to copy over the data to the GPU, we can asynchronously
  // call our kernel. This call will return control to the host immediately and
  // the kernel will run at some point after the previous operation in the
  // stream is complete.
  kernel<<<1, N, 0, stream>>>(dev_a, dev_b, dev_c);

  // copy result.
  hipMemcpyAsync(host_c, dev_c, dataSize, hipMemcpyDeviceToHost, stream);
  // Blocks until stream has completed all operations.
  hipStreamSynchronize(stream);

  // print.
  printf("print A:\n");
  for (int i = 0; i < N; ++i) {
    printf("%d ", host_a[i]);
  }
  printf("\nprint B:\n");
  for (int i = 0; i < N; ++i) {
    printf("%d ", host_b[i]);
  }
  printf("\nprint C:\n");
  for (int i = 0; i < N; ++i) {
    printf("%d ", host_c[i]);
  }
  printf("\n");
  return 0;
}
